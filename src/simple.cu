#include <stdio.h>
#include "hip/hip_runtime.h"
__global__ void simple_kernel(){
    printf("running simple kernel\n");
}

extern "C" void call_simple(){
    printf("before call kernel\n");
    simple_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    printf("after call kernel\n");
}